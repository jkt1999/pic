#include "hip/hip_runtime.h"
/*solves poisson equation with Boltzmann electrons using the Gauss-Seidel scheme*/

#include "PotentialSolver.h"
#include "Field.h"
#include <math.h>
#include <iostream>
#include <stdlib.h>
#include <string.h>
#include <thread>
#include <stdexcept>
#include <mpi.h>
#include "World.h"
#include "hip/hip_runtime.h"
#include ""

using namespace std;

/*container for storing matrix coefficients*/
struct devSeptaD
{
	double *a;
	double *b;
	double *c;
	double *d;
	double *e;
	double *f;
	double *g;
	int nu;
	int ni,nj,nk;

	/*not a destructor to avoid free after copy constructor*/
	void free () {
		hipFree(a);
		hipFree(b);
		hipFree(c);
		hipFree(d);
		hipFree(e);
		hipFree(f);
		hipFree(g);
	}
};

/*builds matrix for system extended by two ghost nodes along each direction*/
void PotentialSolver::initCUDA()
{   
	threads_per_block3.x = 4;
	threads_per_block3.y = 4;
	threads_per_block3.z = 4;

	num_blocks3.x = (A.ni+threads_per_block3.x-1)/threads_per_block3.x;
	num_blocks3.y = (A.nj+threads_per_block3.y-1)/threads_per_block3.y;
	num_blocks3.z = (A.nk+threads_per_block3.z-1)/threads_per_block3.z;
	int  num_blocks = num_blocks3.x*num_blocks3.y*num_blocks3.z;

	int gni = world.ni+2;
	int gnj = world.nj+2;
	int gnk = world.nk+2;
	int gnu = gni*gnj*gnk;
	fvector a(gnu);
	fvector b(gnu);
	fvector c(gnu);
	fvector d(gnu);
	fvector e(gnu);
	fvector f(gnu);
	fvector g(gnu);

	double3 dh = world.getDh();
	float idx = 1.0/dh[0];
	float idy = 1.0/dh[1];
	float idz = 1.0/dh[2];
	float idx2 = idx*idx;	/*1/(dx*dx)*/
	float idy2 = idy*idy;
	float idz2 = idz*idz;

	/*set coefficients, loop over non-ghost nodes*/
	for (int gk=1;gk<gnk-1;gk++)
		for (int gj=1;gj<gnj-1;gj++)
			for (int gi=1;gi<gni-1;gi++)
			{
				int i = gi-1;
				int j = gj-1;
				int k = gk-1;

				int u = gk*(gni*gnj)+gj*gni+gi;

				//dirichlet node?
				if (world.object_id[i][j][k]>0)
				{
					d[u] = 1;
					continue;
				}

				//Neumann boundaries
				if (i==0) {d[u]=idx;c[u]=-idx;}
				else if (i==world.ni-1) {d[u]=idx;e[u]=-idx;}
				else if (j==0) {d[u]=idy;b[u]=-idy;}
				else if (j==world.nj-1) {d[u]=idy;f[u]=-idy;}
				else if (k==0) {d[u]=idz;a[u]=-idz;}
				else if (k==world.nk-1) {d[u]=idz;g[u]=-idz;}
				else {
					//standard internal stencil
					a[u] = idz2; g[u] = idz2;
					b[u] = idy2; f[u] = idy2;
					c[u] = idx2; e[u] = idx2;
					d[u] = -2.0*(idx2+idy2+idz2);
				}
			}

		/*allocate GPU memory for coefficient arrays*/
		hipMalloc((void**)&devA.a, sizeof(float)*gnu);
		hipMalloc((void**)&devA.b, sizeof(float)*gnu);
		hipMalloc((void**)&devA.c, sizeof(float)*gnu);
		hipMalloc((void**)&devA.d, sizeof(float)*gnu);
		hipMalloc((void**)&devA.e, sizeof(float)*gnu);
		hipMalloc((void**)&devA.f, sizeof(float)*gnu);
		hipMalloc((void**)&devA.g, sizeof(float)*gnu);

		/*copy coefficients*/
		hipMemcpy(devA.a,a,sizeof(float)*gnu,hipMemcpyHostToDevice);
		hipMemcpy(devA.b,b,sizeof(float)*gnu,hipMemcpyHostToDevice);
		hipMemcpy(devA.c,c,sizeof(float)*gnu,hipMemcpyHostToDevice);
		hipMemcpy(devA.d,d,sizeof(float)*gnu,hipMemcpyHostToDevice);
		hipMemcpy(devA.e,e,sizeof(float)*gnu,hipMemcpyHostToDevice);
		hipMemcpy(devA.f,f,sizeof(float)*gnu,hipMemcpyHostToDevice);
		hipMemcpy(devA.g,g,sizeof(float)*gnu,hipMemcpyHostToDevice);

		/*copy matrix to

		/*allocate memory for potential and charge density*/
		hipMalloc((void**)&dev_phi, sizeof(float)*gnu);
		hipMalloc((void**)&dev_b, sizeof(float)*gnu);
		hipMalloc((void**)&dev_res, num_blocks*sizeof(float));

		/*allocate CPU memory for res*/
		hipHostAlloc((void**)&host_res,num_blocks*sizeof(float),hipHostMallocDefault);
}


/*Gauss-Seidel Poisson solver*/

/*electron reference parameters*/
__constant__ double dev_n0;
__constant__ double dev_phi0;
__constant__ double dev_kTe0;

/*cuda kernel*/
__global__ void cudaGSupdate (devSeptaD *A, double *phi, double *b, char *object)
{
	int i = blockIdx.x*blockDim.x+threadIdx.x;
	int j = blockIdx.y*blockDim.y+threadIdx.y;
	int k = blockIdx.z*blockDim.z+threadIdx.z;
	int ni = A->ni;
	int nj = A->nj;
	int nk = A->nk;

	/*compute index*/
	int u = k*ni*nj+j*ni+i;
	
	if (i>0 && i<ni-1 &&
		j>0 && j<nj-1 &&
		k>0 && k<nk-1)
	{
		double rhoe = 0;
		
		/*open node*/	
		if (object[u]==0)
			rhoe = (QE*dev_n0*exp((phi[u] - dev_phi0)/dev_kTe0))/EPS_0;

		double g = ((b[u] + rhoe) -
					A->a[u]*phi[u-ni*nj] -
					A->b[u]*phi[u-ni] -
					A->c[u]*phi[u-1] -
					A->e[u]*phi[u+1] -
					A->f[u]*phi[u+ni] -
					A->g[u]*phi[u+ni*nj])/A->d[u];

		/*SOR not converging with Jacobi*/
		phi [u] = g;		
	}
}

/*convergence check*/
__global__ void cudaGSresidue(double *res, devSeptaD *A, double *phi, double *b, char *object)
{
	__shared__ float my_res[1024];		/*1024 is max threads per block*/
	double R = 0;
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int tz = threadIdx.z;
	int i = blockIdx.x*blockDim.x+tx;
	int j = blockIdx.y*blockDim.y+ty;
	int k = blockIdx.z*blockDim.z+tz;
	int ni = A->ni;
	int nj = A->nj;
	int nk = A->nk;

	/*compute index*/
	int u = k*ni*nj+j*ni+i;
	
	if (i>0 && i<ni-1 &&
		j>0 && j<nj-1 &&
		k>0 && k<nk-1)
	{
		double rhoe = 0;
        
        if (object[u]==0)
            rhoe = (QE*dev_n0*exp((phi[u] - dev_phi0)/dev_kTe0))/EPS_0;
						
	   R = (b[u] + rhoe) -
            A->a[u]*phi[u-ni*nj] - 
            A->b[u]*phi[u-ni] - 
            A->c[u]*phi[u-1] - 
            A->d[u]*phi[u] -
            A->e[u]*phi[u+1] - 
            A->f[u]*phi[u+ni] - 
            A->g[u]*phi[u+ni*nj];
	}

	my_res[tz*blockDim.x*blockDim.y+ty*blockDim.x+tx] = R*R;

	/*wait for all threads from block to finish*/
	__syncthreads();

	/*if this is "root", sum up, slow way*/
	if (tx==0 && ty==0 && tz==0)
	{
		double sum = 0;
		for (int i=0;i<blockDim.x*blockDim.y*blockDim.z;i++)
		{
			sum+=my_res[i];
		}
		/*save in global memory*/
		res[blockIdx.z*gridDim.x*gridDim.y+
			blockIdx.y*gridDim.x+
			blockIdx.x] = sum;
	}            
}

/*updated version that leaves potential on the GPU*/
bool PotentialSolver::solveGSCUDA()
{
	bool converged = false;
	double L2;

	/*copy potential on the first time*/
	if (first_time)
	{		
		deflate(phi,world.phi.data);
		CUDA_ERROR(hipMemcpy(dev_phi,phi,A.nu*sizeof(double),hipMemcpyHostToDevice));
		first_time = false;
	}

	/*compute number of blocks for residue checking*/
	int num_blocks = num_blocks3.x*num_blocks3.y*num_blocks3.z;
	 
    /*set RHS to zero on boundary nodes (zero electric field) 
    and to existing potential on fixed nodes        */
	deflate(b,world.rhoi->data);
	
	for (int u=0;u<A.nu;u++)
    {
		if (object[u]<0) b[u] = 0;				/*neumann boundary*/
        else if (object[u]>0) b[u] = phi[u];	/*dirichlet boundary*/
        else b[u] *= -1.0/EPS_0;                /*open node*/
    }
        	
	/*now copy data*/
	CUDA_ERROR(hipMemcpy(dev_b,b,A.nu*sizeof(double),hipMemcpyHostToDevice));
		
	/*solve potential*/
	
	int solver_it;

	//int max_it=25;
	for (solver_it=0;solver_it<max_it;solver_it++)
	{
		/*launch threads*/
		cudaGSupdate<<<num_blocks3,threads_per_block3>>>(dev_devA,dev_phi,dev_b);
	
		if (solver_it%25==0)
		{
			cudaGSresidue<<<num_blocks3,threads_per_block3>>>(dev_res,dev_devA,dev_phi,dev_b,dev_object);
			hipMemcpy(res_pinned,dev_res,num_blocks*sizeof(double),hipMemcpyDeviceToHost);

			double sum=0;
			for (int i=0;i<num_blocks;i++) sum+=res_pinned[i];
		    L2 = sqrt(sum/(A.nu));
			if (L2<tol) {converged=true;break;}
		}
	}

	/*we leave potential on the GPU so don't need to copy back*/
	if (!converged) cerr<<"cudaGS failed to converge, L2 = "<<L2<<endl;
	
    return converged;
}

/********* ADDITIONAL CPU CODE TO SUPPORT CUDA ********************/
/*memory cleanup*/
PotentialSolver::~PotentialSolver()
{
	if (solver_type==GSCUDA)
	{
		devA.free();
		delete(devA);
		CUDA_ERROR(hipFree(dev_devA));
	
		CUDA_ERROR(hipFree(dev_phi));
		CUDA_ERROR(hipFree(dev_b));
		CUDA_ERROR(hipFree(dev_object));
		CUDA_ERROR(hipFree(dev_res));

		/*also free host memory*/
		CUDA_ERROR(hipHostFree(res_pinned));
		delete[] phi;
		delete[] b;	
		CUDA_ERROR(hipFree(world.dev_ef3));
	}

}

/*updates phi on the CPU*/
void PotentialSolver::updateHostPhi()
{
	if (solver_type!=GSCUDA) return;

	hipMemcpy(phi,dev_phi,A.nu*sizeof(double),hipMemcpyDeviceToHost);
	inflate(phi, world.phi.data);
}
