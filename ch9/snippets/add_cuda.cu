#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

//code to run on the GPU
__global__ void add(float a, float b, float *c) {
   *c = a + b;
}

int main(int n_args, char *args[]) 
{
	float *dev_c;
	hipMalloc((void**)&dev_c, sizeof(float));
	add<<<1,1>>>(1,2,dev_c);  //launch add on GPU
	
	float c;
	hipMemcpy(&c,dev_c,sizeof(float),hipMemcpyDeviceToHost);

	cout<<"c = "<<c<<endl;
	return 0;
}
