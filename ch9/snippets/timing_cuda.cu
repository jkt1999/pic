#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
using namespace std;

struct Particle {
	float pos[3] = {0,0,0};
	float vel[3] = {0,0,0};
};

//kernel code to run on the GPU
__global__ void gpu_push(Particle *particles, float dt, size_t N) {
   int p = blockIdx.x*blockDim.x + threadIdx.x;
   if (p<N) {
	   Particle &part = particles[p];
	   for (int i=0;i<3;i++)
		part.pos[i] += part.vel[i]*dt;
	}
}

//code to push a single particle
void push(Particle *part, double dt) {
	for (int i=0;i<3;i++)
		part->pos[i] += part->vel[i]*dt;
}

int main(int n_args, char *args[]) {

  hipFree(0);
  size_t num_particles = 1000000;
 //Particle *particles = new Particle[num_particles];
  Particle *particles;
  hipHostAlloc(&particles,sizeof(Particle)*num_particles,hipHostMallocDefault); //allocate pinned memory

  //set some initial values
  for (size_t i=0;i<num_particles;i++)
	particles[i].vel[0]=1/(double)num_particles;

  const float dt = 0.1;

  //*** CPU particle push ***
   auto start_cpu = chrono::system_clock::now();
   for (size_t i=0;i<num_particles;i++) push(&particles[i],dt);
   auto end_cpu = chrono::system_clock::now();

   //*** GPU particle push ***
  auto start_gpu = chrono::system_clock::now();
  Particle *devParticles;
  hipMalloc((void**)&devParticles, sizeof(Particle)*num_particles);
  hipMemcpy(devParticles,particles,sizeof(Particle)*num_particles,hipMemcpyHostToDevice);

  const int threads_per_block = 1024;
  int num_blocks = (num_particles-1)/threads_per_block + 1;
  cout<<"Creating "<<num_blocks*threads_per_block<<" threads"<<endl;
  gpu_push<<<num_blocks,threads_per_block>>>(devParticles, dt, num_particles);
  hipMemcpy(particles,devParticles,sizeof(Particle)*num_particles,hipMemcpyDeviceToHost);

  auto end_gpu = chrono::system_clock::now();
    
  //output timing info
  std::chrono::duration<double,std::nano> elapsed_cpu = end_cpu - start_cpu;
  std::chrono::duration<double,std::nano> elapsed_gpu = end_gpu - start_gpu;
  cout<<"Time per particle on CPU: "<<elapsed_cpu.count()/num_particles<<" (ns)"<<endl;
  cout<<"Time per particle on GPU: "<<elapsed_gpu.count()/num_particles<<" (ns)"<<endl;

//  delete[] particles;
  hipHostFree(particles);
 
  return 0;
}
